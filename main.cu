#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <hip/hip_runtime.h>
#include "CudaAllocator.h"
#include "hip/hip_runtime_api.h"
#include "ticktock.h"
#include <cmath>
#include <vector>
// #include <thrust/device_vector.h>  // 如果想用 thrust 也是没问题的

// 这是基于“边角料法”的，请把他改成基于“网格跨步循环”的：10 分
template <class Func>
__global__ void fill_sin(int n, Func func) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < n; i += blockDim.x * gridDim.x) {
        func(i);
    }   
}

__global__ void filter_positive(int *counter, int *res, int const *arr, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < n; i += blockDim.x *gridDim.x) {
        if (arr[i] >= 0) {
            // 这里有什么问题？请改正：10 分
            // counter的值不是原子变量，不同线程可能读取相同的counter值从而写入到相同位置
            int loc = atomicAdd(&counter[0], 1);
            res[loc] = arr[i];
        }
    }
}

int main() {
    constexpr int n = 1<<24;
    std::vector<int, CudaAllocator<int>> arr(n);
    std::vector<int, CudaAllocator<int>> res(n);
    std::vector<int, CudaAllocator<int>> counter(1);

    // fill_sin 改成“网格跨步循环”以后，这里三重尖括号里的参数如何调整？10 分
    // 减少block数量让block中的每个线程多运行几次
    TICK(fill_sin);
    fill_sin<<<n / 4096, 1024>>>(n, [arr = arr.data()] __device__ (int i) {
        arr[i] = __sinf(i);
    });
    checkCudaErrors(hipDeviceSynchronize());
    TOCK(fill_sin);

    // 这里的“边角料法”对于不是 1024 整数倍的 n 会出错，为什么？请修复：10 分
    // n向下取整导致末尾的元素未访问
    TICK(fillter_positive);
    filter_positive<<<n / 4096, 512>>>(counter.data(), res.data(), arr.data(), n);
    
    // 这里 CPU 访问数据前漏了一步什么操作？请补上：10 分
    checkCudaErrors(hipDeviceSynchronize());
    TOCK(fillter_positive);

    if (counter[0] <= n / 50) {
        printf("Result too short! %d <= %d\n", counter[0], n / 50);
        return -1;
    }
    for (int i = 0; i < counter[0]; i++) {
        if (res[i] < 0) {
            printf("Wrong At %d: %f < 0\n", i, res[i]);
            return -1;  // 突然想起了ICPC有一年队名叫“蓝翔WA掘机”的，笑不活了:)
        }
    }

    printf("All Correct!\n");  // 还有个队名叫“AC自动机”的，和隔壁“WAWA大哭”对标是吧:)
    return 0;
}
